#include "hip/hip_runtime.h"
#include <cmath>
#include <random>
#include <tuple>
#include <vector>
#include <iostream>

#include <ctc.h>

#include "test.h"

bool small_test() {
    const int alphabet_size = 5;
    const int T = 2;

    std::vector<float> activations = {0.1f, 0.6f, 0.1f, 0.1f, 0.1f,
                                      0.1f, 0.1f, 0.6f, 0.1f, 0.1f};

    // Calculate the score analytically
    float expected_score;
    {
        std::vector<float> probs(activations.size());
        softmax(activations.data(), alphabet_size, T, probs.data());

        // Score calculation is specific to the given activations above
        expected_score = probs[1] * probs[7];
    }

#ifdef __HIPCC__
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#else
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#endif

    float *activations_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#else
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#endif

    std::vector<int> labels = {1, 2};
    std::vector<int> label_lengths = {2};

    std::vector<int> lengths;
    lengths.push_back(T);

    float score;

    ctcOptions options{};
    options.loc = CTC_GPU;
    options.stream = stream;

    size_t gpu_alloc_bytes;
    throw_on_error(get_workspace_size(label_lengths.data(), lengths.data(),
                                      alphabet_size, lengths.size(), options,
                                      &gpu_alloc_bytes),
                   "Error: get_workspace_size in small_test");

    char *ctc_gpu_workspace;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");

#else
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#endif

    throw_on_error(compute_ctc_loss(activations_gpu, nullptr,
                                    labels.data(), label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    &score,
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss in small_test");

    score = std::exp(-score);
    const float eps = 1e-6;

    const float lb = expected_score - eps;
    const float ub = expected_score + eps;

#ifdef __HIPCC__
    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#else
    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#endif

    return (score > lb && score < ub);
}

int offset(int t, int n, int a) {
    constexpr int minibatch = 2;
    constexpr int alphabet_size = 6;
    return (t * minibatch + n) * alphabet_size + a;
}

bool options_test() {
    const int alphabet_size = 6;
    const int T = 5;
    const int minibatch = 2;

    std::vector<float> activations =
        {0.633766f, 0.221185f, 0.0917319f, 0.0129757f, 0.0142857f, 0.0260553f,
         0.30176f, 0.28562f, 0.0831517f, 0.0862751f, 0.0816851f, 0.161508f,

         0.111121f, 0.588392f, 0.278779f, 0.0055756f, 0.00569609f, 0.010436f,
         0.24082f, 0.397533f, 0.0557226f, 0.0546814f, 0.0557528f, 0.19549f,

         0.0357786f, 0.633813f, 0.321418f, 0.00249248f, 0.00272882f, 0.0037688f,
         0.230246f, 0.450868f, 0.0389607f, 0.038309f, 0.0391602f, 0.202456f,

         0.0663296f, 0.643849f, 0.280111f, 0.00283995f, 0.0035545f, 0.00331533f,
         0.280884f, 0.429522f, 0.0326593f, 0.0339046f, 0.0326856f, 0.190345f,

         0.458235f, 0.396634f, 0.123377f, 0.00648837f, 0.00903441f, 0.00623107f,
         0.423286f, 0.315517f, 0.0338439f, 0.0393744f, 0.0339315f, 0.154046f};

    std::vector<float> expected_grads = // from tensorflow
        {-0.366234f, 0.221185f, 0.0917319f, 0.0129757f, 0.0142857f, 0.0260553f,
         -0.69824f, 0.28562f, 0.0831517f, 0.0862751f, 0.0816851f, 0.161508f,

         0.111121f, -0.411608f, 0.278779f, 0.0055756f, 0.00569609f, 0.010436f,
         0.24082f, -0.602467f, 0.0557226f, 0.0546814f, 0.0557528f, 0.19549f,

         0.0357786f, 0.633813f, -0.678582f, 0.00249248f, 0.00272882f, 0.0037688f,
         0.230246f, 0.450868f, 0.0389607f, 0.038309f, 0.0391602f, -0.797544f,

         0.0663296f, -0.356151f, 0.280111f, 0.00283995f, 0.0035545f, 0.00331533f,
         0.280884f, -0.570478f, 0.0326593f, 0.0339046f, 0.0326856f, 0.190345f,

         -0.541765f, 0.396634f, 0.123377f, 0.00648837f, 0.00903441f, 0.00623107f,
         -0.576714f, 0.315517f, 0.0338439f, 0.0393744f, 0.0339315f, 0.154046f};

    // Calculate the expected scores analytically
    auto& a = activations;
    double expected_score[2];
    expected_score[0] =
            -std::log(a[offset(0, 0, 0)] * a[offset(1, 0, 1)] * a[offset(2, 0, 2)]
                     * a[offset(3, 0, 1)] * a[offset(4, 0, 0)]);
    expected_score[1] = 5.42262f; // from tensorflow

    // now take the log to account for the softmax
    for (auto& a : activations) {
        a = std::log(a);
    }

#ifdef __HIPCC__
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#else
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#endif

    float *activations_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#else
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#endif

    std::vector<int> labels = {0, 1, 2, 1, 0,
                               0, 1, 1, 0};

    std::vector<int> label_lengths = {5, 4};

    std::vector<int> lengths = {5, 5};

    float score[2];

    float *grads_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
                   "hipMalloc");
#endif

    ctcOptions options{};
    options.loc = CTC_GPU;
    options.stream = stream;
    options.blank_label = 5;

    size_t gpu_alloc_bytes;
    throw_on_error(get_workspace_size(label_lengths.data(), lengths.data(),
                                      alphabet_size, lengths.size(), options,
                                      &gpu_alloc_bytes),
                   "Error: get_workspace_size in options_test");

    char *ctc_gpu_workspace;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#endif

    throw_on_error(compute_ctc_loss(activations_gpu, grads_gpu,
                                    labels.data(), label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    &score[0],
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss in options_test");

    std::vector<float> grads(alphabet_size * T * minibatch);

#ifdef __HIPCC__
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#else
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#endif

    const double eps = 1e-4;

    bool result = true;
    for (int i = 0; i < grads.size(); i++) {
        const double lb = expected_grads[i] - eps;
        const double ub = expected_grads[i] + eps;
        if (!(grads[i] > lb && grads[i] < ub)) {
            std::cerr << "grad mismatch in options_test"
                      << " expected grad: " << expected_grads[i]
                      << " calculated score: " << grads[i]
                      << " !(" << lb << " < " << grads[i]
                      << " < " << ub << ")" << std::endl;
            result = false;
        }
    }

    for (int i = 0; i < 2; i++) {
        const double lb = expected_score[i] - eps;
        const double ub = expected_score[i] + eps;

        if (!(score[i] > lb && score[i] < ub)) {
            std::cerr << "score mismatch in options_test"
                      << " expected score: " << expected_score[i]
                      << " calculated score: " << score[i] << std::endl;
            result = false;
        }
    }
    return result;
}


bool inf_test() {
    const int alphabet_size = 15;
    const int T = 50;
    const int L = 10;
    const int minibatch = 1;

    std::vector<int> labels = genLabels(alphabet_size, L);
    labels[0] = 2;
    std::vector<int> label_lengths = {L};

    std::vector<float> acts = genActs(alphabet_size * T * minibatch);

    for (int i = 0; i < T; ++i)
        acts[alphabet_size * i + 2] = -1e30;

#ifdef __HIPCC__
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#else
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#endif

    float *acts_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&acts_gpu, acts.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                   acts.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#else
    throw_on_error(hipMalloc(&acts_gpu, acts.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                   acts.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#endif

    std::vector<int> lengths;
    lengths.push_back(T);

    float *grads_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T) * sizeof(float)),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T) * sizeof(float)),
                   "hipMalloc");
#endif

    float cost;

    ctcOptions options{};
    options.loc = CTC_GPU;
    options.stream = stream;

    size_t gpu_alloc_bytes;
    throw_on_error(get_workspace_size(label_lengths.data(), lengths.data(),
                                      alphabet_size, lengths.size(), options,
                                      &gpu_alloc_bytes),
                   "Error: get_workspace_size in inf_test");

    char *ctc_gpu_workspace;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#endif

    throw_on_error(compute_ctc_loss(acts_gpu, grads_gpu,
                                    labels.data(), label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    &cost,
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss in inf_test");

    bool status = std::isinf(cost);

    std::vector<float> grads(alphabet_size * T);

#ifdef __HIPCC__
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");
#else
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");
#endif

    for (int i = 0; i < alphabet_size * T; ++i)
        status &= !std::isnan(grads[i]);

#ifdef __HIPCC__
    throw_on_error(hipFree(acts_gpu), "hipFree");
    throw_on_error(hipFree(grads_gpu), "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace), "hipFree");
    throw_on_error(hipStreamDestroy(stream), "hipStreamDestroy");
#else
    throw_on_error(hipFree(acts_gpu), "hipFree");
    throw_on_error(hipFree(grads_gpu), "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace), "hipFree");
    throw_on_error(hipStreamDestroy(stream), "hipStreamDestroy");
#endif

    return status;
}

float grad_check(int T, int alphabet_size,
                  std::vector<float>& acts,
                  const std::vector<std::vector<int>>& labels,
                  const std::vector<int>& lengths) {

    float epsilon = 1e-2;

    const int minibatch = labels.size();

#ifdef __HIPCC__
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#else
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#endif

    float *acts_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&acts_gpu, acts.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                   acts.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#else
    throw_on_error(hipMalloc(&acts_gpu, acts.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                   acts.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#endif

    std::vector<int> flat_labels;
    std::vector<int> label_lengths;
    for (const auto& l : labels) {
        flat_labels.insert(flat_labels.end(), l.begin(), l.end());
        label_lengths.push_back(l.size());
    }

    std::vector<float> costs(minibatch);

    float *grads_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&grads_gpu, acts.size() * sizeof(float)),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&grads_gpu, acts.size() * sizeof(float)),
                   "hipMalloc");
#endif

    ctcOptions options{};
    options.loc = CTC_GPU;
    options.stream = stream;

    size_t gpu_alloc_bytes;
    throw_on_error(get_workspace_size(label_lengths.data(),
                                      lengths.data(),
                                      alphabet_size,
                                      lengths.size(),
                                      options,
                                      &gpu_alloc_bytes),
                   "Error: get_workspace_size in grad_check");

    char *ctc_gpu_workspace;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#endif

    throw_on_error(compute_ctc_loss(acts_gpu, grads_gpu,
                                    flat_labels.data(),
                                    label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    minibatch,
                                    costs.data(),
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss (0) in grad_check");

    std::vector<float> grads(acts.size());

#ifdef __HIPCC__
    throw_on_error(hipMemcpyAsync(grads.data(),
                                   grads_gpu, grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");
#else
    throw_on_error(hipMemcpyAsync(grads.data(),
                                   grads_gpu, grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");
#endif

    std::vector<float> num_grad(grads.size());

    //perform 2nd order central differencing
    for (int i = 0; i < T * alphabet_size * minibatch; ++i) {
        acts[i] += epsilon;

#ifdef __HIPCC__
        throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                       acts.size() * sizeof(float),
                                       hipMemcpyHostToDevice, stream),
                       "hipMemcpyAsync");
#else
        throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                       acts.size() * sizeof(float),
                                       hipMemcpyHostToDevice, stream),
                       "hipMemcpyAsync");
#endif

        std::vector<float> costsP1(minibatch);
        std::vector<float> costsP2(minibatch);

        throw_on_error(compute_ctc_loss(acts_gpu, NULL,
                                        flat_labels.data(),
                                        label_lengths.data(),
                                        lengths.data(),
                                        alphabet_size,
                                        minibatch,
                                        costsP1.data(),
                                        ctc_gpu_workspace,
                                        options),
                       "Error: compute_ctc_loss (1) in grad_check");

        acts[i] -= 2 * epsilon;

#ifdef __HIPCC__
        throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                       acts.size() * sizeof(float),
                                       hipMemcpyHostToDevice, stream),
                       "hipMemcpyAsync");
#else
        throw_on_error(hipMemcpyAsync(acts_gpu, acts.data(),
                                       acts.size() * sizeof(float),
                                       hipMemcpyHostToDevice, stream),
                       "hipMemcpyAsync");
#endif

        throw_on_error(compute_ctc_loss(acts_gpu, NULL,
                                        flat_labels.data(),
                                        label_lengths.data(),
                                        lengths.data(),
                                        alphabet_size,
                                        minibatch,
                                        costsP2.data(),
                                        ctc_gpu_workspace,
                                        options),
                       "Error: compute_ctc_loss (2) in grad_check");

        float costP1 = std::accumulate(costsP1.begin(), costsP1.end(), 0.);
        float costP2 = std::accumulate(costsP2.begin(), costsP2.end(), 0.);

        acts[i] += epsilon;

        num_grad[i] = (costP1 - costP2) / (2 * epsilon);
    }

    float diff = rel_diff(grads, num_grad);

#ifdef __HIPCC__
    throw_on_error(hipFree(acts_gpu),
                   "hipFree");
    throw_on_error(hipFree(grads_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#else
    throw_on_error(hipFree(acts_gpu),
                   "hipFree");
    throw_on_error(hipFree(grads_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#endif

    return diff;
}

bool run_tests() {
    std::vector<std::tuple<int, int, int, int, float>> problem_sizes =
        { std::make_tuple(28, 50, 15, 1, 1e-5) };

    bool status = true;
    for (auto problem : problem_sizes) {
        int alphabet_size, T, L, minibatch;
        float tol;
        std::tie(alphabet_size, T, L, minibatch, tol) = problem;

        std::vector<float> acts = genActs(alphabet_size * T * minibatch);

        std::vector<std::vector<int>> labels;
        std::vector<int> sizes;
        for (int mb = 0; mb < minibatch; ++mb) {
            int actual_length = L;
            labels.push_back(genLabels(alphabet_size, actual_length));
            sizes.push_back(T);
        }

        float diff = grad_check(T, alphabet_size, acts, labels, sizes);
        status &= (diff < tol);
    }

    return status;
}

bool use_softmax_test() {
    const int alphabet_size = 4;
    const int T = 2;
    const int minibatch = 2;

    std::vector<float> activations = 
        {0.633766f, 0.221185f, 0.0917319f, 0.0129757f,
         0.30176f, 0.28562f, 0.0831517f, 0.0862751f,
                                      
         0.111121f, 0.588392f, 0.278779f, 0.0055756f,
         0.24082f, 0.397533f, 0.0557226f, 0.0546814f};

    std::vector<float> expected_grads = // from tensorflow
        {1.884695f, 0.247554f, 1.096071f, 1.013060f,
         1.352237f, 1.330587f, 0.086707f, 1.090106f,

         1.117530f, 1.801090f, 0.321515f, 1.005591f,
         1.272292f, 0.488149f, 1.057304f, 1.056204f};

    std::vector<double> expected_scores(2);
    auto& a = activations;
    expected_scores[0] = -0.499964f; // from tensorflow
    expected_scores[1] = -0.480685f; // from tensorflow

#ifdef __HIPCC__
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#else
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#endif

    float *activations_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#else
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#endif

    std::vector<int> labels = {1, 2, 2, 1};
    std::vector<int> label_lengths = {2, 2};

    std::vector<int> lengths = {2, 2};
    std::vector<float> costs(minibatch);
    float *grads_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
                   "hipMalloc");
#endif

    ctcOptions options{};
    options.loc = CTC_GPU;
    options.stream = stream;
    options.use_softmax = false;

    size_t gpu_alloc_bytes;
    throw_on_error(get_workspace_size(label_lengths.data(), lengths.data(),
                                      alphabet_size, lengths.size(), options,
                                      &gpu_alloc_bytes),
                   "Error: get_workspace_size in small_test");

    char *ctc_gpu_workspace;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");

#else
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#endif

    throw_on_error(compute_ctc_loss(activations_gpu, grads_gpu,
                                    labels.data(), label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    costs.data(),
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss in small_test");

    std::vector<float> grads(alphabet_size * T * minibatch);

#ifdef __HIPCC__
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#else
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#endif
    const double eps = 1e-4;

    bool result = true;
    for (int i = 0; i < grads.size(); i++) {
        const double lb = expected_grads[i] - eps;
        const double ub = expected_grads[i] + eps;
        if (!(grads[i] > lb && grads[i] < ub)) {
            std::cerr << "grad mismatch in use_softmax_test"
                    << " expected grad: " << expected_grads[i]
                    << " calculated score: " << grads[i]
                    << " !(" << lb << " < " << grads[i]
                    << " < " << ub << ")" << std::endl;
            result = false;
        }
    }

    for (int i = 0; i < 2; i++) {
        const double lb = expected_scores[i] - eps;
        const double ub = expected_scores[i] + eps;

        if (!(costs[i] > lb && costs[i] < ub)) {
            std::cerr << "score mismatch in use_softmax_test"
                    << " expected score: " << expected_scores[i]
                    << " calculated score: " << costs[i] << std::endl;
            result = false;
        }
    }
    return result;
}

bool zero_infinity_test() {
    const int alphabet_size = 4;
    const int T = 2;
    const int minibatch = 2;

    std::vector<float> activations = 
        {0.633766f, 0.221185f, 0.0917319f, 0.0129757f,
         0.30176f, -1e30, 0.0831517f, 0.0862751f,
                                     
         0.111121f, 0.588392f, 0.278779f, 0.0055756f,
         0.24082f, -1e30, 0.0557226f, 0.0546814f};

    std::vector<float> expected_grads = // from tensorflow
        {1.884695f, 0.247554f, 1.096071f, 1.013060f,
         0.0f, 0.0f, 0.0f, 0.0f,

         1.117530f, 1.801090f, 0.321515f, 1.005591f,
         0.0f, 0.0f, 0.0f, 0.0f};

    std::vector<double> expected_scores(2);
    auto& a = activations;
    expected_scores[0] = -0.5f; // from tensorflow
    expected_scores[1] = 0.0f;

#ifdef __HIPCC__
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#else
    hipStream_t stream;
    throw_on_error(hipStreamCreate(&stream),
                   "hipStreamCreate");
#endif

    float *activations_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#else
    throw_on_error(hipMalloc(&activations_gpu,
                   activations.size() * sizeof(float)),
                   "hipMalloc");
    throw_on_error(hipMemcpyAsync(activations_gpu, activations.data(),
                                   activations.size() * sizeof(float),
                                   hipMemcpyHostToDevice, stream),
                   "hipMemcpyAsync");
#endif

    std::vector<int> labels = {1, 2, 2, 1};
    std::vector<int> label_lengths = {2, 2};

    std::vector<int> lengths = {2, 2};
    std::vector<float> costs(minibatch);
    float *grads_gpu;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
                   "hipMalloc");
#else
    throw_on_error(hipMalloc(&grads_gpu, (alphabet_size * T * minibatch) * sizeof(float)),
                   "hipMalloc");
#endif

    ctcOptions options{};
    options.loc = CTC_GPU;
    options.stream = stream;
    options.use_softmax = false;
    options.zero_infinity = true;

    size_t gpu_alloc_bytes;
    throw_on_error(get_workspace_size(label_lengths.data(), lengths.data(),
                                      alphabet_size, lengths.size(), options,
                                      &gpu_alloc_bytes),
                   "Error: get_workspace_size in small_test");

    char *ctc_gpu_workspace;

#ifdef __HIPCC__
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");

#else
    throw_on_error(hipMalloc(&ctc_gpu_workspace, gpu_alloc_bytes),
                   "hipMalloc");
#endif

    throw_on_error(compute_ctc_loss(activations_gpu, grads_gpu,
                                    labels.data(), label_lengths.data(),
                                    lengths.data(),
                                    alphabet_size,
                                    lengths.size(),
                                    costs.data(),
                                    ctc_gpu_workspace,
                                    options),
                   "Error: compute_ctc_loss in small_test");

    std::vector<float> grads(alphabet_size * T * minibatch);

#ifdef __HIPCC__
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#else
    throw_on_error(hipMemcpyAsync(grads.data(), grads_gpu,
                                   grads.size() * sizeof(float),
                                   hipMemcpyDeviceToHost, stream),
                   "hipMemcpyAsync");
    throw_on_error(hipStreamSynchronize(stream), "hipStreamSynchronize");

    throw_on_error(hipFree(activations_gpu),
                   "hipFree");
    throw_on_error(hipFree(ctc_gpu_workspace),
                   "hipFree");
    throw_on_error(hipStreamDestroy(stream),
                   "hipStreamDestroy");
#endif
    const double eps = 1e-4;

    bool result = true;
    for (int i = 0; i < grads.size(); i++) {
        const double lb = expected_grads[i] - eps;
        const double ub = expected_grads[i] + eps;
        if (!(grads[i] > lb && grads[i] < ub)) {
            std::cerr << "grad mismatch in zero_infinity_test"
                    << " expected grad: " << expected_grads[i]
                    << " calculated score: " << grads[i]
                    << " !(" << lb << " < " << grads[i]
                    << " < " << ub << ")" << std::endl;
            result = false;
        }
    }

    for (int i = 0; i < 2; i++) {
        const double lb = expected_scores[i] - eps;
        const double ub = expected_scores[i] + eps;

        if (!(costs[i] > lb && costs[i] < ub)) {
            std::cerr << "score mismatch in zero_infinity_test"
                    << " expected score: " << expected_scores[i]
                    << " calculated score: " << costs[i] << std::endl;
            result = false;
        }
    }
    return result;
}

int main(void) {
    if (get_warpctc_version() != 2) {
        std::cerr << "Invalid WarpCTC version." << std::endl;
        return 1;
    }

    std::cout << "Running GPU tests" << std::endl;

#ifdef __HIPCC__
    throw_on_error(hipSetDevice(0), "hipSetDevice");
#else
    throw_on_error(hipSetDevice(0), "hipSetDevice");
#endif

    bool status = true;
    status &= small_test();
    status &= options_test();
    status &= inf_test();
    status &= run_tests();
    status &= use_softmax_test();
    status &= zero_infinity_test();

    if (status) {
        std::cout << "Tests pass" << std::endl;
        return 0;
    } else {
        std::cout << "Some or all tests fail" << std::endl;
        return 1;
    }
}
